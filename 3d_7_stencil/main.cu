
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;
#define IN_3D(_z,_y,_x)   in[(_z)*(M)*(N)+(_y)*(N)+(_x)]
#define OUT_3D(_z,_y,_x) out[(_z)*(M)*(N)+(_y)*(N)+(_x)]

#define SM_Z (8+2)
#define SM_M (4+2)
#define SM_N (8+2)
#define LOC_3D(_z,_y,_x) local[(_z)*(SM_M)*(SM_N)+(_y)*(SM_N)+(_x)]

#define SM_2D_M (4+2)
#define SM_2D_N (64+2)
#define LOC_2D(_y,_x) local[(_y)*(SM_2D_N)+(_x)]
// #define LOC_2D2(_y,_x) local[(_y)*(SM_2D_N2+2*halo)+(_x)]
// #define LOC_L_2D(_z,_y,_x) local[(_z)*(SM_2D_M*SM_2D_N)+(_y)*(SM_2D_N)+(_x)]

#define DATA_TYPE float
#define warpSize 32 

// #define __DEBUG

#ifdef __DEBUG
#define ITER 1
#else
#define ITER 100
#endif


// #define TEMP
#define SPAC1

float GetGFLOPS(int z, int m, int n, int count, int ops, float time) 
{
    float f = (z*m*n)*(float)(ops)*(float)(count)/time * 1.0e-09;
    return f;
}

float GetThroughput(int z, int m, int n, int count, float time) 
{
    return (z*m*n) * sizeof(DATA_TYPE) * 2.0 * ((float)count)
            / time * 1.0e-09;    
          
}

void Init_Input_3D(DATA_TYPE *in, int Z, int M, int N)
{
    srand(time(NULL));

    for(int k = 0; k < Z; k++)
        for(int j = 0; j < M; j++)
            for(int i = 0; i < N; i++)
#ifdef __DEBUG
                IN_3D(k,j,i) = 1; //(DATA_TYPE)rand() * 100.0 / RAND_MAX;
#else
                IN_3D(k,j,i) = (DATA_TYPE)rand()*100.0 / RAND_MAX;
#endif
}

void Clear_Output_3D(DATA_TYPE *in, int Z, int M, int N)
{
    for(int k = 0; k < Z; k++)
        for(int j = 0; j < M; j++)
            for(int i = 0; i < N; i++)
                IN_3D(k,j,i) = 0;
}

void Show_Me(DATA_TYPE *in, int Z, int M, int N, std::string prompt)
{
    std::cout << prompt << std::endl;
    for(int k = 0; k < Z; k++)
    {
        for(int j = 0; j < M; j++)
        {
            for(int i = 0; i < N; i++)
                std::cout << IN_3D(k,j,i) << ",";
            std::cout << std::endl;
        }
        std::cout << std::endl;
    }
}

void Stencil_Seq(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, 
        DATA_TYPE a2, DATA_TYPE a3, DATA_TYPE a4, DATA_TYPE a5, DATA_TYPE a6, 
        int Z, int M, int N)
{
    for(int k = 0; k < Z; k++)
    {
        int b = (k == 0)      ? k : k - 1;
        int t = (k == Z-1)    ? k : k + 1;
        for(int j = 0; j < M; j++)
        {
            int n = (j == 0)      ? j : j - 1;
            int s = (j == M-1)    ? j : j + 1;
            for(int i = 0; i < N; i++)
            {
                int w = (i == 0)      ? i : i - 1;
                int e = (i == N-1)    ? i : i + 1;
                OUT_3D(k,j,i) = a0 * IN_3D(b  ,j  ,i  ) +
                                a1 * IN_3D(k  ,n  ,i  ) +
                                a2 * IN_3D(k  ,j  ,w  ) +
                                a3 * IN_3D(k  ,j  ,i  ) +
                                a4 * IN_3D(k  ,j  ,e  ) +
                                a5 * IN_3D(k  ,s  ,i  ) +
                                a6 * IN_3D(t  ,j  ,i  ) ;
            }
        }
    }
}

inline double tol_finder(int error_tol)
{
    double val = 1.0;
    for(; error_tol > 0; error_tol--)
        val *= 10;
    return 1.0/(double)val;
}

bool Verify(DATA_TYPE *test, DATA_TYPE *ref, int n)
{
    bool flag = true;
    double precision = tol_finder(2);

    for(int i = 0; i < n; i++)
    {
        if(fabs(test[i]-ref[i]) > precision)
        {
            std::cout << "difference: " << fabs(test[i]-ref[i])-precision << std::endl;
            std::cout << "wrong at " << i << " test:" << test[i] << " (ref: " << ref[i] << ")";
            std::cout << std::endl;
            flag = false;
            break;
        }
    }
    return flag;
}

__global__ void Stencil_Cuda(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, 
        DATA_TYPE a2, DATA_TYPE a3, DATA_TYPE a4, DATA_TYPE a5, DATA_TYPE a6, 
        int Z, int M, int N) 
{
    int i = threadIdx.x + blockIdx.x * blockDim.x ;
    int j = threadIdx.y + blockIdx.y * blockDim.y ;
    int k = threadIdx.z + blockIdx.z * blockDim.z ;

    int b = (k == 0)      ? k : k - 1;
    int t = (k == Z-1)    ? k : k + 1;
    int n = (j == 0)      ? j : j - 1;
    int s = (j == M-1)    ? j : j + 1;
    int w = (i == 0)      ? i : i - 1;
    int e = (i == N-1)    ? i : i + 1;
    OUT_3D(k,j,i) = a0 * IN_3D(b  ,j  ,i  ) +
                    a1 * IN_3D(k  ,n  ,i  ) +
                    a2 * IN_3D(k  ,j  ,w  ) +
                    a3 * IN_3D(k  ,j  ,i  ) +
                    a4 * IN_3D(k  ,j  ,e  ) +
                    a5 * IN_3D(k  ,s  ,i  ) +
                    a6 * IN_3D(t  ,j  ,i  ) ;
}

__global__ void Stencil_Cuda_Sweep(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, 
        DATA_TYPE a2, DATA_TYPE a3, DATA_TYPE a4, DATA_TYPE a5, DATA_TYPE a6, 
        int Z, int M, int N) 
{
    int i = threadIdx.x + blockIdx.x * blockDim.x ;
    int j = threadIdx.y + blockIdx.y * blockDim.y ;

    const int block_z = Z / gridDim.z;
    int k = block_z * blockIdx.z ;
    const int k_end = k + block_z;

    int n = (j == 0)      ? j : j - 1;
    int s = (j == M-1)    ? j : j + 1;
    int w = (i == 0)      ? i : i - 1;
    int e = (i == N-1)    ? i : i + 1;
#pragma unroll // it seems the loop-unroll is useless to performance
    for(; k < k_end; ++k)
    {
        int b = (k == 0)      ? k : k - 1;
        int t = (k == Z-1)    ? k : k + 1;
        OUT_3D(k,j,i) = a0 * IN_3D(b  ,j  ,i  ) +
                        a1 * IN_3D(k  ,n  ,i  ) +
                        a2 * IN_3D(k  ,j  ,w  ) +
                        a3 * IN_3D(k  ,j  ,i  ) +
                        a4 * IN_3D(k  ,j  ,e  ) +
                        a5 * IN_3D(k  ,s  ,i  ) +
                        a6 * IN_3D(t  ,j  ,i  ) ;
    }
}

__global__ void Stencil_Cuda_Sweep_Sm(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, 
        DATA_TYPE a2, DATA_TYPE a3, DATA_TYPE a4, DATA_TYPE a5, DATA_TYPE a6, 
        int Z, int M, int N)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x ;
    int j = threadIdx.y + blockIdx.y * blockDim.y ;
    const int block_z = Z / gridDim.z;
    int k = block_z * blockIdx.z ;
    const int k_end = k + block_z;
    int li = threadIdx.x + 1;
    int lj = threadIdx.y + 1;

    extern __shared__ DATA_TYPE local[];

    DATA_TYPE t1, t2, t3;
    DATA_TYPE r1, r2, r3, r4;
    DATA_TYPE sum = 0.0;
    int n = (j == 0)      ? j : j - 1;
    int s = (j == M-1)    ? j : j + 1;
    int w = (i == 0)      ? i : i - 1;
    int e = (i == N-1)    ? i : i + 1;
    // load current layer
    t3 = IN_3D(k, j, i); 
    if(threadIdx.x == 0)            r1 = IN_3D(k,j,w);
    if(threadIdx.x == blockDim.x-1) r2 = IN_3D(k,j,e);
    if(threadIdx.y == 0)            r3 = IN_3D(k,n,i);
    if(threadIdx.y == blockDim.y-1) r4 = IN_3D(k,s,i);

    // load previous layer (same with k)
    t2 = IN_3D(k, j, i);

#pragma unroll // it seems the loop-unroll is useless to performance
    for(; k < k_end; ++k)
    {
        sum = 0.0;
        t1 = t2;
        t2 = t3;
        LOC_2D(lj,li) = t2;
        if(threadIdx.x == 0)            LOC_2D(lj  ,li-1) = r1; 
        if(threadIdx.x == blockDim.x-1) LOC_2D(lj  ,li+1) = r2; 
        if(threadIdx.y == 0)            LOC_2D(lj-1,li  ) = r3; 
        if(threadIdx.y == blockDim.y-1) LOC_2D(lj+1,li  ) = r4; 

        // load next layer
        int t = (k == Z-1)    ? k : k+1;
        t3 = IN_3D(t, j, i); 
        if(threadIdx.x == 0)            r1 = IN_3D(t,j,w);
        if(threadIdx.x == blockDim.x-1) r2 = IN_3D(t,j,e);
        if(threadIdx.y == 0)            r3 = IN_3D(t,n,i);
        if(threadIdx.y == blockDim.y-1) r4 = IN_3D(t,s,i);

        sum += a0 * t1 + a3 * t2 + a5 * t3;
        __syncthreads();
        sum += a1 * LOC_2D(lj-1,li  );
        sum += a2 * LOC_2D(lj  ,li-1);
        sum += a4 * LOC_2D(lj  ,li+1);
        sum += a6 * LOC_2D(lj+1,li  );

        OUT_3D(k,j,i) = sum;
        __syncthreads();
    }
}

__global__ void Stencil_Cuda_Sm(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, 
        DATA_TYPE a2, DATA_TYPE a3, DATA_TYPE a4, DATA_TYPE a5, DATA_TYPE a6, 
        int Z, int M, int N) 
{
    int i = threadIdx.x + blockIdx.x * blockDim.x ;
    int j = threadIdx.y + blockIdx.y * blockDim.y ;
    int k = threadIdx.z + blockIdx.z * blockDim.z ;

    int li = threadIdx.x + 1;
    int lj = threadIdx.y + 1;
    int lk = threadIdx.z + 1;

    __shared__ DATA_TYPE local[SM_Z*SM_M*SM_N];

    LOC_3D(lk,lj,li) = IN_3D(k,j,i);

    int b = (k == 0)      ? k : k - 1;
    int t = (k == Z-1)    ? k : k + 1;
    int n = (j == 0)      ? j : j - 1;
    int s = (j == M-1)    ? j : j + 1;
    int w = (i == 0)      ? i : i - 1;
    int e = (i == N-1)    ? i : i + 1;
    if(threadIdx.x == 0)            LOC_3D(lk  ,lj  ,li-1) = IN_3D(k,j,w);
    if(threadIdx.x == blockDim.x-1) LOC_3D(lk  ,lj  ,li+1) = IN_3D(k,j,e);
    if(threadIdx.y == 0)            LOC_3D(lk  ,lj-1,li  ) = IN_3D(k,n,i);
    if(threadIdx.y == blockDim.y-1) LOC_3D(lk  ,lj+1,li  ) = IN_3D(k,s,i);
    if(threadIdx.z == 0)            LOC_3D(lk-1,lj  ,li  ) = IN_3D(b,j,i);
    if(threadIdx.z == blockDim.z-1) LOC_3D(lk+1,lj  ,li  ) = IN_3D(t,j,i);
    __syncthreads();

    OUT_3D(k,j,i) = a0 * LOC_3D(lk-1,lj  ,li  ) +
                    a1 * LOC_3D(lk  ,lj-1,li  ) +
                    a2 * LOC_3D(lk  ,lj  ,li-1) +
                    a3 * LOC_3D(lk  ,lj  ,li  ) +
                    a4 * LOC_3D(lk  ,lj  ,li+1) +
                    a5 * LOC_3D(lk  ,lj+1,li  ) +
                    a6 * LOC_3D(lk+1,lj  ,li  ) ;
}

__global__ void Stencil_Cuda_Shfl(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, 
        DATA_TYPE a2, DATA_TYPE a3, DATA_TYPE a4, DATA_TYPE a5, DATA_TYPE a6, 
        int Z, int M, int N)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x ;
    int j = threadIdx.y + blockIdx.y * blockDim.y ;
    int k = threadIdx.z + blockIdx.z * blockDim.z ;
    int tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.y * blockDim.x;
    int lane_id = tid % warpSize;
    int warp_id_x = (threadIdx.x + blockIdx.x * blockDim.x)>>3; // because the warp dimensions are 
    int warp_id_y = (threadIdx.y + blockIdx.y * blockDim.y)>>2; // 1x4x8, warp_ids are division of 
    int warp_id_z = (threadIdx.z + blockIdx.z * blockDim.z)>>0; // there numbers
    int new_i = (warp_id_x<<3) + lane_id%10 - 1;     // 10 is extended dimension of i
    int new_j = (warp_id_y<<2) + (lane_id/10)%6 - 1; // 6  is extended dimension of j 
    int new_k = (warp_id_z<<0) + lane_id/60 - 1;     // 60 is extended area of ixj = 10x6
    DATA_TYPE threadInput0, threadInput1, threadInput2, threadInput3, threadInput4, threadInput5;
    
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput0 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+32)%10 -1;
    new_j = (warp_id_y<<2) + ((lane_id+32)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+32)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput1 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+64)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+64)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+64)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput2 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+96)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+96)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+96)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput3 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+128)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+128)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+128)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput4 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+160)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+160)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+160)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput5 = IN_3D(new_k, new_j, new_i);

    DATA_TYPE sum = 0.0;
    int friend_id;
    DATA_TYPE tx, ty, tz;
    friend_id = (lane_id+11+((lane_id>>3)<<1))&(warpSize-1);
    tx = a0*__shfl(threadInput0, friend_id);
    ty = a0*__shfl(threadInput1, friend_id);
    sum += (lane_id < 17)? tx: ty;

    friend_id = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
    tx = a1*__shfl(threadInput1, friend_id);
    ty = a1*__shfl(threadInput2, friend_id);
    tz = a1*__shfl(threadInput3, friend_id);
    sum += (lane_id < 3 )? tx: ((lane_id < 29)? ty: tz);

    friend_id = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    tx = a2*__shfl(threadInput2, friend_id);
    ty = a2*__shfl(threadInput3, friend_id);
    sum += (lane_id < 22)? tx: ty;

    friend_id = (lane_id+7 +((lane_id>>3)<<1))&(warpSize-1);
    tx = a3*__shfl(threadInput2, friend_id);
    ty = a3*__shfl(threadInput3, friend_id);
    sum += (lane_id < 21)? tx: ty;

    friend_id = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    tx = a4*__shfl(threadInput2, friend_id);
    ty = a4*__shfl(threadInput3, friend_id);
    sum += (lane_id < 20)? tx: ty;

    friend_id = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    tx = a5*__shfl(threadInput2, friend_id);
    ty = a5*__shfl(threadInput3, friend_id);
    sum += (lane_id < 13)? tx: ty;

    friend_id = (lane_id+3 +((lane_id>>3)<<1))&(warpSize-1);
    tx = a6*__shfl(threadInput4, friend_id);
    ty = a6*__shfl(threadInput5, friend_id);
    sum += (lane_id < 24)? tx: ty;

    OUT_3D(k,j,i) = sum;

}

__global__ void Stencil_Cuda_Shfl2(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, 
        DATA_TYPE a2, DATA_TYPE a3, DATA_TYPE a4, DATA_TYPE a5, DATA_TYPE a6, 
        int Z, int M, int N)
{
    int tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.y * blockDim.x;
    int lane_id = tid % warpSize;

    int i = threadIdx.x + blockIdx.x * blockDim.x ;
    int j = threadIdx.y + blockIdx.y * blockDim.y ;
    int k = (((threadIdx.z + blockIdx.z * blockDim.z)>>0)<<1) + (lane_id>>5) ; 
    // thread coarsening: related to warp dimensions 1x4x8. 
    // We coarsen from dimension z from 2^0 to 2^1, also need to know there are how many values in dimension z

    int warp_id_x = (threadIdx.x + blockIdx.x * blockDim.x)>>3; // because the warp dimensions are 
    int warp_id_y = (threadIdx.y + blockIdx.y * blockDim.y)>>2; // 1x4x8, warp_ids are division of 
    int warp_id_z = (((threadIdx.z + blockIdx.z * blockDim.z)>>0)<<1) + (lane_id>>5); // these numbers
    int new_i = (warp_id_x<<3) + lane_id%10-1;     // 10 is extended dimension of i
    int new_j = (warp_id_y<<2) + (lane_id/10)%6-1; // 6  is extended dimension of j 
    int new_k = (warp_id_z<<0) + lane_id/60-1;     // 60 is extended area of ixj = 10x6
    DATA_TYPE threadInput0, threadInput1, threadInput2, threadInput3, threadInput4, threadInput5,
              threadInput6, threadInput7;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput0 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+32)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+32)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+32)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput1 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+64)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+64)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+64)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput2 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+96)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+96)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+96)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput3 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+128)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+128)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+128)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput4 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+160)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+160)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+160)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput5 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+192)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+192)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+192)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput6 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+224)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+224)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+224)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput7 = IN_3D(new_k, new_j, new_i);

    DATA_TYPE sum0 = 0.0;
    DATA_TYPE sum1 = 0.0;
    int friend_id0, friend_id1;
    DATA_TYPE tx0, ty0, tz0, tx1, ty1, tz1;
    friend_id0 = (lane_id+11+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+7 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = a0*__shfl(threadInput0, friend_id0);
    ty0 = a0*__shfl(threadInput1, friend_id0);
    tx1 = a0*__shfl(threadInput2, friend_id1);
    ty1 = a0*__shfl(threadInput3, friend_id1);
    sum0 += (lane_id < 17)? tx0: ty0;
    sum1 += (lane_id < 21)? tx1: ty1;

    friend_id0 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = a1*__shfl(threadInput1, friend_id0);
    ty0 = a1*__shfl(threadInput2, friend_id0);
    tz0 = a1*__shfl(threadInput3, friend_id0);
    tx1 = a1*__shfl(threadInput3, friend_id1);
    ty1 = a1*__shfl(threadInput4, friend_id1);
    sum0 += (lane_id < 3 )? tx0: ((lane_id < 29)? ty0: tz0);
    sum1 += (lane_id < 7 )? tx1: ty1;

    friend_id0 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = a2*__shfl(threadInput2, friend_id0);
    ty0 = a2*__shfl(threadInput3, friend_id0);
    tx1 = a2*__shfl(threadInput4, friend_id1);
    ty1 = a2*__shfl(threadInput5, friend_id1);
    sum0 += (lane_id < 22)? tx0: ty0;
    sum1 += (lane_id < 24)? tx1: ty1;

    friend_id0 = (lane_id+7 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+3 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = a3*__shfl(threadInput2, friend_id0);
    ty0 = a3*__shfl(threadInput3, friend_id0);
    tx1 = a3*__shfl(threadInput4, friend_id1);
    ty1 = a3*__shfl(threadInput5, friend_id1);
    sum0 += (lane_id < 21)? tx0: ty0;
    sum1 += (lane_id < 24)? tx1: ty1;

    friend_id0 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = a4*__shfl(threadInput2, friend_id0);
    ty0 = a4*__shfl(threadInput3, friend_id0);
    tx1 = a4*__shfl(threadInput4, friend_id1);
    ty1 = a4*__shfl(threadInput5, friend_id1);
    sum0 += (lane_id < 20)? tx0: ty0;
    sum1 += (lane_id < 24)? tx1: ty1;

    friend_id0 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = a5*__shfl(threadInput2, friend_id0);
    ty0 = a5*__shfl(threadInput3, friend_id0);
    tx1 = a5*__shfl(threadInput4, friend_id1);
    ty1 = a5*__shfl(threadInput5, friend_id1);
    sum0 += (lane_id < 13)? tx0: ty0;
    sum1 += (lane_id < 16)? tx1: ty1;

    friend_id0 = (lane_id+3 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+31+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = a6*__shfl(threadInput4, friend_id0);
    ty0 = a6*__shfl(threadInput5, friend_id0);
    tx1 = a6*__shfl(threadInput5, friend_id1);
    ty1 = a6*__shfl(threadInput6, friend_id1);
    tz1 = a6*__shfl(threadInput7, friend_id1);
    sum0 += (lane_id < 24)? tx0: ty0;
    sum1 += (lane_id < 1 )? tx1: ((lane_id < 27)? ty1: tz1);

    OUT_3D(k,j,i) = sum0;
    OUT_3D(k+1,j,i) = sum1;

}

__global__ void Stencil_Cuda_Shfl4(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, 
        DATA_TYPE a2, DATA_TYPE a3, DATA_TYPE a4, DATA_TYPE a5, DATA_TYPE a6, 
        int Z, int M, int N)
{
    int tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.y * blockDim.x;
    int lane_id = tid % warpSize;

    int i = threadIdx.x + blockIdx.x * blockDim.x ;
    int j = threadIdx.y + blockIdx.y * blockDim.y ;
    int k = (((threadIdx.z + blockIdx.z * blockDim.z)>>0)<<2) + (lane_id>>5) ; 
    // Thread coarsening: related to warp dimensions 1x4x8. 
    // We coarsen from dimension z from 2^0 to 2^2, also need to know there are how many values in dimension z,
    // which is (lane_id>>5) 

    int warp_id_x = (threadIdx.x + blockIdx.x * blockDim.x)>>3; // because the warp dimensions are 
    int warp_id_y = (threadIdx.y + blockIdx.y * blockDim.y)>>2; // 1x4x8, warp_ids are division of 
    int warp_id_z = (((threadIdx.z + blockIdx.z * blockDim.z)>>0)<<2) + (lane_id>>5); // these numbers
    int new_i = (warp_id_x<<3) + lane_id%10-1;     // 10 is extended dimension of i
    int new_j = (warp_id_y<<2) + (lane_id/10)%6-1; // 6  is extended dimension of j 
    int new_k = (warp_id_z<<0) + lane_id/60-1;     // 60 is extended area of ixj = 10x6
    DATA_TYPE threadInput0, threadInput1, threadInput2, threadInput3, threadInput4, threadInput5,
              threadInput6, threadInput7, threadInput8, threadInput9, threadInput10; //, threadInput11;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput0 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+32)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+32)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+32)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput1 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+64)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+64)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+64)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput2 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+96)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+96)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+96)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput3 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+128)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+128)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+128)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput4 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+160)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+160)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+160)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput5 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+192)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+192)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+192)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput6 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+224)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+224)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+224)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput7 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+256)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+256)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+256)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput8 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+288)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+288)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+288)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput9 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+320)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+320)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+320)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput10 = IN_3D(new_k, new_j, new_i);
    // new_i = (warp_id_x<<3) + (lane_id+352)%10-1;
    // new_j = (warp_id_y<<2) + ((lane_id+352)/10)%6-1;
    // new_k = (warp_id_z<<0) + (lane_id+352)/60-1;
    // new_k = (new_k == -1)   ? 0   : new_k;
    // new_k = (new_k >= Z)    ? Z-1 : new_k;
    // new_j = (new_j == -1)   ? 0   : new_j;
    // new_j = (new_j >= M)    ? M-1 : new_j;
    // new_i = (new_i == -1)   ? 0   : new_i;
    // new_i = (new_i >= N)    ? N-1 : new_i;
    // threadInput11 = IN_3D(new_k, new_j, new_i);

    DATA_TYPE sum0 = 0.0;
    DATA_TYPE sum1 = 0.0;
    DATA_TYPE sum2 = 0.0;
    DATA_TYPE sum3 = 0.0;
    int friend_id0, friend_id1, friend_id2, friend_id3;
    DATA_TYPE tx0, ty0, tz0, tx1, ty1, tz1, tx2, ty2, tz2, tx3, ty3, tz3;
    friend_id0 = (lane_id+11+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+7 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+3 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+31+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = a0*__shfl(threadInput0, friend_id0);
    ty0 = a0*__shfl(threadInput1, friend_id0);
    tx1 = a0*__shfl(threadInput2, friend_id1);
    ty1 = a0*__shfl(threadInput3, friend_id1);
    tx2 = a0*__shfl(threadInput4, friend_id2);
    ty2 = a0*__shfl(threadInput5, friend_id2);
    tx3 = a0*__shfl(threadInput5, friend_id3);
    ty3 = a0*__shfl(threadInput6, friend_id3);
    tz3 = a0*__shfl(threadInput7, friend_id3);
    sum0 += (lane_id < 17)? tx0: ty0;
    sum1 += (lane_id < 21)? tx1: ty1;
    sum2 += (lane_id < 24)? tx2: ty2;
    sum3 += (lane_id < 1 )? tx3: ((lane_id < 27)? ty3: tz3);

    friend_id0 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = a1*__shfl(threadInput1, friend_id0);
    ty0 = a1*__shfl(threadInput2, friend_id0);
    tz0 = a1*__shfl(threadInput3, friend_id0);
    tx1 = a1*__shfl(threadInput3, friend_id1);
    ty1 = a1*__shfl(threadInput4, friend_id1);
    tx2 = a1*__shfl(threadInput5, friend_id2);
    ty2 = a1*__shfl(threadInput6, friend_id2);
    tx3 = a1*__shfl(threadInput7, friend_id3);
    ty3 = a1*__shfl(threadInput8, friend_id3);
    sum0 += (lane_id < 3 )? tx0: ((lane_id < 29)? ty0: tz0);
    sum1 += (lane_id < 7 )? tx1: ty1;
    sum2 += (lane_id < 9 )? tx2: ty2;
    sum3 += (lane_id < 13)? tx3: ty3;

    friend_id0 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = a2*__shfl(threadInput2, friend_id0);
    ty0 = a2*__shfl(threadInput3, friend_id0);
    tx1 = a2*__shfl(threadInput4, friend_id1);
    ty1 = a2*__shfl(threadInput5, friend_id1);
    tx2 = a2*__shfl(threadInput5, friend_id2);
    ty2 = a2*__shfl(threadInput6, friend_id2);
    tz2 = a2*__shfl(threadInput7, friend_id2);
    tx3 = a2*__shfl(threadInput7, friend_id3);
    ty3 = a2*__shfl(threadInput8, friend_id3);
    sum0 += (lane_id < 22)? tx0: ty0;
    sum1 += (lane_id < 24)? tx1: ty1;
    sum2 += (lane_id < 2 )? tx2: ((lane_id < 28)? ty2: tz2);
    sum3 += (lane_id < 6 )? tx3: ty3;

    friend_id0 = (lane_id+7 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+3 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+31+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+27+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = a3*__shfl(threadInput2, friend_id0);
    ty0 = a3*__shfl(threadInput3, friend_id0);
    tx1 = a3*__shfl(threadInput4, friend_id1);
    ty1 = a3*__shfl(threadInput5, friend_id1);
    tx2 = a3*__shfl(threadInput5, friend_id2);
    ty2 = a3*__shfl(threadInput6, friend_id2);
    tz2 = a3*__shfl(threadInput7, friend_id2);
    tx3 = a3*__shfl(threadInput7, friend_id3);
    ty3 = a3*__shfl(threadInput8, friend_id3);
    tz3 = a3*__shfl(threadInput9, friend_id3);
    sum0 += (lane_id < 21)? tx0: ty0;
    sum1 += (lane_id < 24)? tx1: ty1;
    sum2 += (lane_id < 1 )? tx2: ((lane_id < 27)? ty2: tz2);
    sum3 += (lane_id < 5 )? tx3: ((lane_id < 31)? ty3: tz3);

    friend_id0 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = a4*__shfl(threadInput2, friend_id0);
    ty0 = a4*__shfl(threadInput3, friend_id0);
    tx1 = a4*__shfl(threadInput4, friend_id1);
    ty1 = a4*__shfl(threadInput5, friend_id1);
    tx2 = a4*__shfl(threadInput6, friend_id2);
    ty2 = a4*__shfl(threadInput7, friend_id2);
    tx3 = a4*__shfl(threadInput7, friend_id3);
    ty3 = a4*__shfl(threadInput8, friend_id3);
    tz3 = a4*__shfl(threadInput9, friend_id3);
    sum0 += (lane_id < 20)? tx0: ty0;
    sum1 += (lane_id < 24)? tx1: ty1;
    sum2 += (lane_id < 26)? tx2: ty2;
    sum3 += (lane_id < 4 )? tx3: ((lane_id < 30)? ty3: tz3);

    friend_id0 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+9 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+5 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = a5*__shfl(threadInput2, friend_id0);
    ty0 = a5*__shfl(threadInput3, friend_id0);
    tx1 = a5*__shfl(threadInput4, friend_id1);
    ty1 = a5*__shfl(threadInput5, friend_id1);
    tx2 = a5*__shfl(threadInput6, friend_id2);
    ty2 = a5*__shfl(threadInput7, friend_id2);
    tx3 = a5*__shfl(threadInput8, friend_id3);
    ty3 = a5*__shfl(threadInput9, friend_id3);
    sum0 += (lane_id < 13)? tx0: ty0;
    sum1 += (lane_id < 16)? tx1: ty1;
    sum2 += (lane_id < 19)? tx2: ty2;
    sum3 += (lane_id < 23)? tx3: ty3;

    friend_id0 = (lane_id+3 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+31+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+27+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+23+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = a6*__shfl(threadInput4 , friend_id0);
    ty0 = a6*__shfl(threadInput5 , friend_id0);
    tx1 = a6*__shfl(threadInput5 , friend_id1);
    ty1 = a6*__shfl(threadInput6 , friend_id1);
    tz1 = a6*__shfl(threadInput7 , friend_id1);
    tx2 = a6*__shfl(threadInput7 , friend_id2);
    ty2 = a6*__shfl(threadInput8 , friend_id2);
    tz2 = a6*__shfl(threadInput9 , friend_id2);
    tx3 = a6*__shfl(threadInput9 , friend_id3);
    ty3 = a6*__shfl(threadInput10, friend_id3);
    sum0 += (lane_id < 24)? tx0: ty0;
    sum1 += (lane_id < 1 )? tx1: ((lane_id < 27)? ty1: tz1);
    sum2 += (lane_id < 5 )? tx2: ((lane_id < 31)? ty2: tz2);
    sum3 += (lane_id < 8 )? tx3: ty3;

    OUT_3D(k,j,i) = sum0;
    OUT_3D(k+1,j,i) = sum1;
    OUT_3D(k+2,j,i) = sum2;
    OUT_3D(k+3,j,i) = sum3;
}

__global__ void Stencil_Cuda_Shfl8(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, 
        DATA_TYPE a2, DATA_TYPE a3, DATA_TYPE a4, DATA_TYPE a5, DATA_TYPE a6, 
        int Z, int M, int N)
{
    int tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.y * blockDim.x;
    int lane_id = tid % warpSize;

    int i = threadIdx.x + blockIdx.x * blockDim.x ;
    int j = threadIdx.y + blockIdx.y * blockDim.y ;
    int k = (((threadIdx.z + blockIdx.z * blockDim.z)>>0)<<3) + (lane_id>>5) ; 
    // Thread coarsening: related to warp dimensions 1x4x8. 
    // We coarsen from dimension z from 2^0 to 2^3, also need to know there are how many values in dimension z,
    // which is (lane_id>>5) 

    int warp_id_x = (threadIdx.x + blockIdx.x * blockDim.x)>>3; // because the warp dimensions are 
    int warp_id_y = (threadIdx.y + blockIdx.y * blockDim.y)>>2; // 1x4x8, warp_ids are division of 
    int warp_id_z = (((threadIdx.z + blockIdx.z * blockDim.z)>>0)<<3) + (lane_id>>5); // these numbers
    int new_i = (warp_id_x<<3) + lane_id%10-1;     // 10 is extended dimension of i
    int new_j = (warp_id_y<<2) + (lane_id/10)%6-1; // 6  is extended dimension of j 
    int new_k = (warp_id_z<<0) + lane_id/60-1;     // 60 is extended area of ixj = 10x6
    DATA_TYPE threadInput0, threadInput1, threadInput2, threadInput3, threadInput4, threadInput5,
              threadInput6, threadInput7, threadInput8, threadInput9, threadInput10, threadInput11,
              threadInput12, threadInput13, threadInput14, threadInput15, threadInput16, threadInput17,
              threadInput18;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput0 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+32)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+32)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+32)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput1 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+64)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+64)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+64)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput2 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+96)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+96)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+96)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput3 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+128)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+128)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+128)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput4 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+160)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+160)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+160)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput5 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+192)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+192)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+192)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput6 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+224)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+224)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+224)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput7 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+256)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+256)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+256)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput8 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+288)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+288)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+288)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput9 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+320)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+320)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+320)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput10 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+352)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+352)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+352)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput11 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+384)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+384)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+384)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput12 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+416)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+416)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+416)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput13 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+448)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+448)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+448)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput14 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+480)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+480)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+480)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput15 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+512)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+512)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+512)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput16 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+544)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+544)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+544)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput17 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+576)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+576)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+576)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput18 = IN_3D(new_k, new_j, new_i);

    DATA_TYPE sum0 = 0.0;
    DATA_TYPE sum1 = 0.0;
    DATA_TYPE sum2 = 0.0;
    DATA_TYPE sum3 = 0.0;
    DATA_TYPE sum4 = 0.0;
    DATA_TYPE sum5 = 0.0;
    DATA_TYPE sum6 = 0.0;
    DATA_TYPE sum7 = 0.0;
    int friend_id0, friend_id1, friend_id2, friend_id3;
    int friend_id4, friend_id5, friend_id6, friend_id7;
    DATA_TYPE tx0, ty0, tz0, tx1, ty1, tz1, tx2, ty2, tz2, tx3, ty3, tz3;
    DATA_TYPE rx0, ry0, rz0, rx1, ry1, rz1, rx2, ry2, rz2, rx3, ry3, rz3;
    friend_id0 = (lane_id+11+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+7 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+3 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+31+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+27+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+23+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+19+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+15+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = a0*__shfl(threadInput0, friend_id0);
    ty0 = a0*__shfl(threadInput1, friend_id0);
    tx1 = a0*__shfl(threadInput2, friend_id1);
    ty1 = a0*__shfl(threadInput3, friend_id1);
    tx2 = a0*__shfl(threadInput4, friend_id2);
    ty2 = a0*__shfl(threadInput5, friend_id2);
    tx3 = a0*__shfl(threadInput5, friend_id3);
    ty3 = a0*__shfl(threadInput6, friend_id3);
    tz3 = a0*__shfl(threadInput7, friend_id3);
    rx0 = a0*__shfl(threadInput7, friend_id4);
    ry0 = a0*__shfl(threadInput8, friend_id4);
    rz0 = a0*__shfl(threadInput9, friend_id4);
    rx1 = a0*__shfl(threadInput9, friend_id5);
    ry1 = a0*__shfl(threadInput10, friend_id5);
    rx2 = a0*__shfl(threadInput11, friend_id6);
    ry2 = a0*__shfl(threadInput12, friend_id6);
    rx3 = a0*__shfl(threadInput13, friend_id7);
    ry3 = a0*__shfl(threadInput14, friend_id7);

    sum0 += (lane_id < 17)? tx0: ty0;
    sum1 += (lane_id < 21)? tx1: ty1;
    sum2 += (lane_id < 24)? tx2: ty2;
    sum3 += (lane_id < 1 )? tx3: ((lane_id < 27)? ty3: tz3);
    sum4 += (lane_id < 5 )? rx0: ((lane_id < 31)? ry0: rz0);
    sum5 += (lane_id < 8 )? rx1: ry1;
    sum6 += (lane_id < 11)? rx2: ry2;
    sum7 += (lane_id < 15)? rx3: ry3;

    friend_id0 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+9 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+5 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = a1*__shfl(threadInput1, friend_id0);
    ty0 = a1*__shfl(threadInput2, friend_id0);
    tz0 = a1*__shfl(threadInput3, friend_id0);
    tx1 = a1*__shfl(threadInput3, friend_id1);
    ty1 = a1*__shfl(threadInput4, friend_id1);
    tx2 = a1*__shfl(threadInput5, friend_id2);
    ty2 = a1*__shfl(threadInput6, friend_id2);
    tx3 = a1*__shfl(threadInput7, friend_id3);
    ty3 = a1*__shfl(threadInput8, friend_id3);
    rx0 = a1*__shfl(threadInput9, friend_id4);
    ry0 = a1*__shfl(threadInput10, friend_id4);
    rx1 = a1*__shfl(threadInput11, friend_id5);
    ry1 = a1*__shfl(threadInput12, friend_id5);
    rx2 = a1*__shfl(threadInput13, friend_id6);
    ry2 = a1*__shfl(threadInput14, friend_id6);
    rx3 = a1*__shfl(threadInput15, friend_id7);
    ry3 = a1*__shfl(threadInput16, friend_id7);
    sum0 += (lane_id < 3 )? tx0: ((lane_id < 29)? ty0: tz0);
    sum1 += (lane_id < 7 )? tx1: ty1;
    sum2 += (lane_id < 9 )? tx2: ty2;
    sum3 += (lane_id < 13)? tx3: ty3;
    sum4 += (lane_id < 16)? rx0: ry0;
    sum5 += (lane_id < 19)? rx1: ry1;
    sum6 += (lane_id < 23)? rx2: ry2;
    sum7 += (lane_id < 25)? rx3: ry3;

    friend_id0 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = a2*__shfl(threadInput2, friend_id0);
    ty0 = a2*__shfl(threadInput3, friend_id0);
    tx1 = a2*__shfl(threadInput4, friend_id1);
    ty1 = a2*__shfl(threadInput5, friend_id1);
    tx2 = a2*__shfl(threadInput5, friend_id2);
    ty2 = a2*__shfl(threadInput6, friend_id2);
    tz2 = a2*__shfl(threadInput7, friend_id2);
    tx3 = a2*__shfl(threadInput7, friend_id3);
    ty3 = a2*__shfl(threadInput8, friend_id3);
    rx0 = a2*__shfl(threadInput9, friend_id4);
    ry0 = a2*__shfl(threadInput10, friend_id4);
    rx1 = a2*__shfl(threadInput11, friend_id5);
    ry1 = a2*__shfl(threadInput12, friend_id5);
    rx2 = a2*__shfl(threadInput13, friend_id6);
    ry2 = a2*__shfl(threadInput14, friend_id6);
    rx3 = a2*__shfl(threadInput15, friend_id7);
    ry3 = a2*__shfl(threadInput16, friend_id7);
    sum0 += (lane_id < 22)? tx0: ty0;
    sum1 += (lane_id < 24)? tx1: ty1;
    sum2 += (lane_id < 2 )? tx2: ((lane_id < 28)? ty2: tz2);
    sum3 += (lane_id < 6 )? tx3: ty3;
    sum4 += (lane_id < 8 )? rx0: ry0;
    sum5 += (lane_id < 12)? rx1: ry1;
    sum6 += (lane_id < 16)? rx2: ry2;
    sum7 += (lane_id < 18)? rx3: ry3;

    friend_id0 = (lane_id+7 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+3 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+31+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+27+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+23+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+19+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+15+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+11+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = a3*__shfl(threadInput2, friend_id0);
    ty0 = a3*__shfl(threadInput3, friend_id0);
    tx1 = a3*__shfl(threadInput4, friend_id1);
    ty1 = a3*__shfl(threadInput5, friend_id1);
    tx2 = a3*__shfl(threadInput5, friend_id2);
    ty2 = a3*__shfl(threadInput6, friend_id2);
    tz2 = a3*__shfl(threadInput7, friend_id2);
    tx3 = a3*__shfl(threadInput7, friend_id3);
    ty3 = a3*__shfl(threadInput8, friend_id3);
    tz3 = a3*__shfl(threadInput9, friend_id3);
    rx0 = a3*__shfl(threadInput9, friend_id4);
    ry0 = a3*__shfl(threadInput10, friend_id4);
    rx1 = a3*__shfl(threadInput11, friend_id5);
    ry1 = a3*__shfl(threadInput12, friend_id5);
    rx2 = a3*__shfl(threadInput13, friend_id6);
    ry2 = a3*__shfl(threadInput14, friend_id6);
    rx3 = a3*__shfl(threadInput15, friend_id7);
    ry3 = a3*__shfl(threadInput16, friend_id7);
    sum0 += (lane_id < 21)? tx0: ty0;
    sum1 += (lane_id < 24)? tx1: ty1;
    sum2 += (lane_id < 1 )? tx2: ((lane_id < 27)? ty2: tz2);
    sum3 += (lane_id < 5 )? tx3: ((lane_id < 31)? ty3: tz3);
    sum4 += (lane_id < 8 )? rx0: ry0;
    sum5 += (lane_id < 11)? rx1: ry1;
    sum6 += (lane_id < 15)? rx2: ry2;
    sum7 += (lane_id < 17)? rx3: ry3;

    friend_id0 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = a4*__shfl(threadInput2, friend_id0);
    ty0 = a4*__shfl(threadInput3, friend_id0);
    tx1 = a4*__shfl(threadInput4, friend_id1);
    ty1 = a4*__shfl(threadInput5, friend_id1);
    tx2 = a4*__shfl(threadInput6, friend_id2);
    ty2 = a4*__shfl(threadInput7, friend_id2);
    tx3 = a4*__shfl(threadInput7, friend_id3);
    ty3 = a4*__shfl(threadInput8, friend_id3);
    tz3 = a4*__shfl(threadInput9, friend_id3);
    rx0 = a4*__shfl(threadInput9, friend_id4);
    ry0 = a4*__shfl(threadInput10, friend_id4);
    rx1 = a4*__shfl(threadInput11, friend_id5);
    ry1 = a4*__shfl(threadInput12, friend_id5);
    rx2 = a4*__shfl(threadInput13, friend_id6);
    ry2 = a4*__shfl(threadInput14, friend_id6);
    rx3 = a4*__shfl(threadInput15, friend_id7);
    ry3 = a4*__shfl(threadInput16, friend_id7);
    sum0 += (lane_id < 20)? tx0: ty0;
    sum1 += (lane_id < 24)? tx1: ty1;
    sum2 += (lane_id < 26)? tx2: ty2;
    sum3 += (lane_id < 4 )? tx3: ((lane_id < 30)? ty3: tz3);
    sum4 += (lane_id < 8 )? rx0: ry0;
    sum5 += (lane_id < 10)? rx1: ry1;
    sum6 += (lane_id < 14)? rx2: ry2;
    sum7 += (lane_id < 16)? rx3: ry3;

    friend_id0 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+9 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+5 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = a5*__shfl(threadInput2, friend_id0);
    ty0 = a5*__shfl(threadInput3, friend_id0);
    tx1 = a5*__shfl(threadInput4, friend_id1);
    ty1 = a5*__shfl(threadInput5, friend_id1);
    tx2 = a5*__shfl(threadInput6, friend_id2);
    ty2 = a5*__shfl(threadInput7, friend_id2);
    tx3 = a5*__shfl(threadInput8, friend_id3);
    ty3 = a5*__shfl(threadInput9, friend_id3);
    rx0 = a5*__shfl(threadInput10, friend_id4);
    ry0 = a5*__shfl(threadInput11, friend_id4);
    rx1 = a5*__shfl(threadInput11, friend_id5);
    ry1 = a5*__shfl(threadInput12, friend_id5);
    rz1 = a5*__shfl(threadInput13, friend_id5);
    rx2 = a5*__shfl(threadInput13, friend_id6);
    ry2 = a5*__shfl(threadInput14, friend_id6);
    rx3 = a5*__shfl(threadInput15, friend_id7);
    ry3 = a5*__shfl(threadInput16, friend_id7);
    sum0 += (lane_id < 13)? tx0: ty0;
    sum1 += (lane_id < 16)? tx1: ty1;
    sum2 += (lane_id < 19)? tx2: ty2;
    sum3 += (lane_id < 23)? tx3: ty3;
    sum4 += (lane_id < 25)? rx0: ry0;
    sum5 += (lane_id < 3 )? rx1: ((lane_id < 29)? ry1: rz1);
    sum6 += (lane_id < 7 )? rx2: ry2;
    sum7 += (lane_id < 9 )? rx3: ry3;

    friend_id0 = (lane_id+3 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+31+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+27+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+23+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+19+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+15+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+11+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+7 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = a6*__shfl(threadInput4 , friend_id0);
    ty0 = a6*__shfl(threadInput5 , friend_id0);
    tx1 = a6*__shfl(threadInput5 , friend_id1);
    ty1 = a6*__shfl(threadInput6 , friend_id1);
    tz1 = a6*__shfl(threadInput7 , friend_id1);
    tx2 = a6*__shfl(threadInput7 , friend_id2);
    ty2 = a6*__shfl(threadInput8 , friend_id2);
    tz2 = a6*__shfl(threadInput9 , friend_id2);
    tx3 = a6*__shfl(threadInput9 , friend_id3);
    ty3 = a6*__shfl(threadInput10, friend_id3);
    rx0 = a6*__shfl(threadInput11, friend_id4);
    ry0 = a6*__shfl(threadInput12, friend_id4);
    rx1 = a6*__shfl(threadInput13, friend_id5);
    ry1 = a6*__shfl(threadInput14, friend_id5);
    rx2 = a6*__shfl(threadInput15, friend_id6);
    ry2 = a6*__shfl(threadInput16, friend_id6);
    rx3 = a6*__shfl(threadInput17, friend_id7);
    ry3 = a6*__shfl(threadInput18, friend_id7);
    sum0 += (lane_id < 24)? tx0: ty0;
    sum1 += (lane_id < 1 )? tx1: ((lane_id < 27)? ty1: tz1);
    sum2 += (lane_id < 5 )? tx2: ((lane_id < 31)? ty2: tz2);
    sum3 += (lane_id < 8 )? tx3: ty3;
    sum4 += (lane_id < 11)? rx0: ry0;
    sum5 += (lane_id < 15)? rx1: ry1;
    sum6 += (lane_id < 17)? rx2: ry2;
    sum7 += (lane_id < 21)? rx3: ry3;

    OUT_3D(k,j,i) = sum0;
    OUT_3D(k+1,j,i) = sum1;
    OUT_3D(k+2,j,i) = sum2;
    OUT_3D(k+3,j,i) = sum3;
    OUT_3D(k+4,j,i) = sum4;
    OUT_3D(k+5,j,i) = sum5;
    OUT_3D(k+6,j,i) = sum6;
    OUT_3D(k+7,j,i) = sum7;
}

__global__ void Stencil_Cuda_Sweep_Shfl(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, 
        DATA_TYPE a2, DATA_TYPE a3, DATA_TYPE a4, DATA_TYPE a5, DATA_TYPE a6, 
        int Z, int M, int N)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x ;
    int j = threadIdx.y + blockIdx.y * blockDim.y ;
    const int block_z = Z / gridDim.z;
    int k = block_z * blockIdx.z ;
    const int k_end = k + block_z;
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    int lane_id = tid % warpSize;
    int warp_id_x = (threadIdx.x + blockIdx.x * blockDim.x)>>3; // because the warp dimensions are 
    int warp_id_y = (threadIdx.y + blockIdx.y * blockDim.y)>>2; // 1x4x8, warp_ids are division of 
    DATA_TYPE tx, ty;
    int friend_id;
    int new_i, new_j;
    DATA_TYPE t3_threadInput0, t3_threadInput1;
    DATA_TYPE t2_threadInput0, t2_threadInput1;
    DATA_TYPE t1_threadInput0, t1_threadInput1;

#define SM_2D_M2 32 
#define SM_2D_N2 8 

    DATA_TYPE sum = 0.0;

    int b = (k == 0)      ? k : k - 1;
    // t3 is current layer; t2 is previous layer
    new_i = (warp_id_x<<3) + lane_id%10-1;     // 10 is extended dimension of i
    new_j = (warp_id_y<<2) + lane_id/10-1;     
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    t3_threadInput0 = IN_3D(k  , new_j, new_i);
    t2_threadInput0 = IN_3D(b  , new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+32)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+32)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    t3_threadInput1 = IN_3D(k  , new_j, new_i);
    t2_threadInput1 = IN_3D(b  , new_j, new_i);

#pragma unroll // it seems the loop-unroll is useless to performance
    for(; k < k_end; ++k)
    {
        sum = 0.0;
        // move the current storage down 
        t1_threadInput0 = t2_threadInput0;
        t1_threadInput1 = t2_threadInput1;
        t2_threadInput0 = t3_threadInput0;
        t2_threadInput1 = t3_threadInput1;

        int t = (k == Z-1)    ? k : k + 1;
        new_i = (warp_id_x<<3) + lane_id%10-1;  
        new_j = (warp_id_y<<2) + lane_id/10-1;     
        new_j = (new_j == -1)   ? 0   : new_j;
        new_j = (new_j >= M)    ? M-1 : new_j;
        new_i = (new_i == -1)   ? 0   : new_i;
        new_i = (new_i >= N)    ? N-1 : new_i;
        t3_threadInput0 = IN_3D(t, new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+32)%10-1;
        new_j = (warp_id_y<<2) + (lane_id+32)/10-1;
        new_j = (new_j == -1)   ? 0   : new_j;
        new_j = (new_j >= M)    ? M-1 : new_j;
        new_i = (new_i == -1)   ? 0   : new_i;
        new_i = (new_i >= N)    ? N-1 : new_i;
        t3_threadInput1 = IN_3D(t, new_j, new_i);

        friend_id = (lane_id+11+((lane_id>>3)<<1))&(warpSize-1);
        tx = a0*__shfl(t1_threadInput0, friend_id);
        ty = a0*__shfl(t1_threadInput1, friend_id);
        sum += (lane_id < 17)? tx: ty;
        tx = a3*__shfl(t2_threadInput0, friend_id);
        ty = a3*__shfl(t2_threadInput1, friend_id);
        sum += (lane_id < 17)? tx: ty;
        tx = a5*__shfl(t3_threadInput0, friend_id);
        ty = a5*__shfl(t3_threadInput1, friend_id);
        sum += (lane_id < 17)? tx: ty;
        friend_id = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
        tx = a1*__shfl(t2_threadInput0, friend_id);
        ty = a1*__shfl(t2_threadInput1, friend_id);
        sum += (lane_id < 25)? tx: ty;
        friend_id = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
        tx = a2*__shfl(t2_threadInput0, friend_id);
        ty = a2*__shfl(t2_threadInput1, friend_id);
        sum += (lane_id < 18)? tx: ty;
        friend_id = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
        tx = a4*__shfl(t2_threadInput0, friend_id);
        ty = a4*__shfl(t2_threadInput1, friend_id);
        sum += (lane_id < 16)? tx: ty;
        friend_id = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
        tx = a6*__shfl(t2_threadInput0, friend_id);
        ty = a6*__shfl(t2_threadInput1, friend_id);
        sum += (lane_id < 9 )? tx: ty;

        OUT_3D(k,j,i) = sum;
    }
}

__global__ void Stencil_Cuda_Sweep_Shfl2(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, 
        DATA_TYPE a2, DATA_TYPE a3, DATA_TYPE a4, DATA_TYPE a5, DATA_TYPE a6, 
        int Z, int M, int N)
{
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    int lane_id = tid % warpSize;

    int i = threadIdx.x + blockIdx.x * blockDim.x ;
    int j = (((threadIdx.y + blockIdx.y * blockDim.y)>>2)<<3) + (lane_id>>3) ;

    const int block_z = Z / gridDim.z;
    int k = block_z * blockIdx.z ;
    const int k_end = k + block_z;
    int warp_id_x = (threadIdx.x + blockIdx.x * blockDim.x)>>3; // because the warp dimensions are 
    int warp_id_y = ((((threadIdx.y + blockIdx.y * blockDim.y)>>2)<<3) + (lane_id>>3))>>2; // 1x4x8, warp_ids are division of 
    DATA_TYPE tx0, ty0;
    DATA_TYPE tx1, ty1, tz1;
    int friend_id0, friend_id1;
    int new_i, new_j;
    DATA_TYPE t3_threadInput0, t3_threadInput1, t3_threadInput2, t3_threadInput3;
    DATA_TYPE t2_threadInput0, t2_threadInput1, t2_threadInput2, t2_threadInput3;
    DATA_TYPE t1_threadInput0, t1_threadInput1, t1_threadInput2, t1_threadInput3;

#define SM_2D_M2 32 
#define SM_2D_N2 8 

    DATA_TYPE sum0 = 0.0;
    DATA_TYPE sum1 = 0.0;

    // t3 is current layer; t2 is previous layer
    int b = (k == 0)      ? k : k - 1;
    new_i = (warp_id_x<<3) + lane_id%10-1;     // 10 is extended dimension of i
    new_j = (warp_id_y<<2) + lane_id/10-1;     
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    t3_threadInput0 = IN_3D(k  , new_j, new_i);
    t2_threadInput0 = IN_3D(b  , new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+32)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+32)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    t3_threadInput1 = IN_3D(k  , new_j, new_i);
    t2_threadInput1 = IN_3D(b  , new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+64)%10;
    new_j = (warp_id_y<<2) + (lane_id+64)/10;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    t3_threadInput2 = IN_3D(k  , new_j, new_i);
    t2_threadInput2 = IN_3D(b  , new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+96)%10;
    new_j = (warp_id_y<<2) + (lane_id+96)/10;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    t3_threadInput3 = IN_3D(k  , new_j, new_i);
    t2_threadInput3 = IN_3D(b  , new_j, new_i);

#pragma unroll // it seems the loop-unroll is useless to performance
    for(; k < k_end; ++k)
    {
        sum0 = 0.0;
        sum1 = 0.0;
        // move the current storage down 
        t1_threadInput0 = t2_threadInput0;
        t1_threadInput1 = t2_threadInput1;
        t1_threadInput2 = t2_threadInput2;
        t1_threadInput3 = t2_threadInput3;
        t2_threadInput0 = t3_threadInput0;
        t2_threadInput1 = t3_threadInput1;
        t2_threadInput2 = t3_threadInput2;
        t2_threadInput3 = t3_threadInput3;

        int t = (k == Z-1)    ? k : k + 1;
        new_i = (warp_id_x<<3) + lane_id%10-1;  
        new_j = (warp_id_y<<2) + lane_id/10-1;     
        new_j = (new_j == -1)   ? 0   : new_j;
        new_j = (new_j >= M)    ? M-1 : new_j;
        new_i = (new_i == -1)   ? 0   : new_i;
        new_i = (new_i >= N)    ? N-1 : new_i;
        t3_threadInput0 = IN_3D(t  , new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+32)%10-1;
        new_j = (warp_id_y<<2) + (lane_id+32)/10-1;
        new_j = (new_j == -1)   ? 0   : new_j;
        new_j = (new_j >= M)    ? M-1 : new_j;
        new_i = (new_i == -1)   ? 0   : new_i;
        new_i = (new_i >= N)    ? N-1 : new_i;
        t3_threadInput1 = IN_3D(t  , new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+64)%10-1;
        new_j = (warp_id_y<<2) + (lane_id+64)/10-1;
        new_j = (new_j == -1)   ? 0   : new_j;
        new_j = (new_j >= M)    ? M-1 : new_j;
        new_i = (new_i == -1)   ? 0   : new_i;
        new_i = (new_i >= N)    ? N-1 : new_i;
        t3_threadInput2 = IN_3D(t  , new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+96)%10-1;
        new_j = (warp_id_y<<2) + (lane_id+96)/10-1;
        new_j = (new_j == -1)   ? 0   : new_j;
        new_j = (new_j >= M)    ? M-1 : new_j;
        new_i = (new_i == -1)   ? 0   : new_i;
        new_i = (new_i >= N)    ? N-1 : new_i;
        t3_threadInput3 = IN_3D(t  , new_j, new_i);

        friend_id0 = (lane_id+11+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+19+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = a0*__shfl(t1_threadInput0, friend_id0);
        ty0 = a0*__shfl(t1_threadInput1, friend_id0);
        tx1 = a0*__shfl(t1_threadInput1, friend_id1);
        ty1 = a0*__shfl(t1_threadInput2, friend_id1);
        sum0 += (lane_id < 17)? tx0: ty0;
        sum1 += (lane_id < 11)? tx1: ty1;
        tx0 = a3*__shfl(t2_threadInput0, friend_id0);
        ty0 = a3*__shfl(t2_threadInput1, friend_id0);
        tx1 = a3*__shfl(t2_threadInput1, friend_id1);
        ty1 = a3*__shfl(t2_threadInput2, friend_id1);
        sum0 += (lane_id < 17)? tx0: ty0;
        sum1 += (lane_id < 11)? tx1: ty1;
        tx0 = a5*__shfl(t3_threadInput0, friend_id0);
        ty0 = a5*__shfl(t3_threadInput1, friend_id0);
        tx1 = a5*__shfl(t3_threadInput1, friend_id1);
        ty1 = a5*__shfl(t3_threadInput2, friend_id1);
        sum0 += (lane_id < 17)? tx0: ty0;
        sum1 += (lane_id < 11)? tx1: ty1;

        friend_id0 = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+9 +((lane_id>>3)<<1))&(warpSize-1);
        tx0 = a1*__shfl(t2_threadInput0, friend_id0);
        ty0 = a1*__shfl(t2_threadInput1, friend_id0);
        tx1 = a1*__shfl(t2_threadInput1, friend_id1);
        ty1 = a1*__shfl(t2_threadInput2, friend_id1);
        sum0 += (lane_id < 25)? tx0: ty0;
        sum1 += (lane_id < 19)? tx1: ty1;
        friend_id0 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = a2*__shfl(t2_threadInput0, friend_id0);
        ty0 = a2*__shfl(t2_threadInput1, friend_id0);
        tx1 = a2*__shfl(t2_threadInput1, friend_id1);
        ty1 = a2*__shfl(t2_threadInput2, friend_id1);
        sum0 += (lane_id < 18)? tx0: ty0;
        sum1 += (lane_id < 12)? tx1: ty1;
        friend_id0 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = a4*__shfl(t2_threadInput0, friend_id0);
        ty0 = a4*__shfl(t2_threadInput1, friend_id0);
        tx1 = a4*__shfl(t2_threadInput1, friend_id1);
        ty1 = a4*__shfl(t2_threadInput2, friend_id1);
        sum0 += (lane_id < 16)? tx0: ty0;
        sum1 += (lane_id < 10)? tx1: ty1;
        friend_id0 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = a5*__shfl(t2_threadInput0, friend_id0);
        ty0 = a5*__shfl(t2_threadInput1, friend_id0);
        tx1 = a5*__shfl(t2_threadInput1, friend_id1);
        ty1 = a5*__shfl(t2_threadInput2, friend_id1);
        tz1 = a5*__shfl(t2_threadInput3, friend_id1);
        sum0 += (lane_id < 9 )? tx0: ty0;
        sum1 += (lane_id < 3 )? tx1: ((lane_id < 29)? ty1: tz1);

        OUT_3D(k,j  ,i) = sum0;
        OUT_3D(k,j+4,i) = sum1;
    }
}

__global__ void Stencil_Cuda_Sweep_Shfl4(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, 
        DATA_TYPE a2, DATA_TYPE a3, DATA_TYPE a4, DATA_TYPE a5, DATA_TYPE a6, 
        int Z, int M, int N)
{
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    int lane_id = tid % warpSize;

    int i = threadIdx.x + blockIdx.x * blockDim.x ;
    int j = (((threadIdx.y + blockIdx.y * blockDim.y)>>2)<<4) + (lane_id>>3) ;

    const int block_z = Z / gridDim.z;
    int k = block_z * blockIdx.z ;
    const int k_end = k + block_z;
    int warp_id_x = (threadIdx.x + blockIdx.x * blockDim.x)>>3; // because the warp dimensions are 
    int warp_id_y = ((((threadIdx.y + blockIdx.y * blockDim.y)>>2)<<4) + (lane_id>>3))>>2; // 1x4x8, warp_ids are division of 
    DATA_TYPE tx0, ty0;
    DATA_TYPE tx1, ty1, tz1;
    DATA_TYPE tx2, ty2, tz2;
    DATA_TYPE tx3, ty3, tz3;
    int friend_id0, friend_id1;
    int friend_id2, friend_id3;
    int new_i, new_j;
    DATA_TYPE t3_threadInput0, t3_threadInput1, t3_threadInput2, t3_threadInput3, t3_threadInput4, t3_threadInput5;
    DATA_TYPE t2_threadInput0, t2_threadInput1, t2_threadInput2, t2_threadInput3, t2_threadInput4, t2_threadInput5;
    DATA_TYPE t1_threadInput0, t1_threadInput1, t1_threadInput2, t1_threadInput3, t1_threadInput4, t1_threadInput5;

#define SM_2D_M2 32 
#define SM_2D_N2 8 

    DATA_TYPE sum0 = 0.0;
    DATA_TYPE sum1 = 0.0;
    DATA_TYPE sum2 = 0.0;
    DATA_TYPE sum3 = 0.0;

    // t3 is current layer; t2 is previous layer
    int b = (k == 0)      ? k : k - 1;
    new_i = (warp_id_x<<3) + lane_id%10-1;     // 10 is extended dimension of i
    new_j = (warp_id_y<<2) + lane_id/10-1;     
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    t3_threadInput0 = IN_3D(k  , new_j, new_i);
    t2_threadInput0 = IN_3D(b  , new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+32)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+32)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    t3_threadInput1 = IN_3D(k  , new_j, new_i);
    t2_threadInput1 = IN_3D(b  , new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+64)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+64)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    t3_threadInput2 = IN_3D(k  , new_j, new_i);
    t2_threadInput2 = IN_3D(b  , new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+96)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+96)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    t3_threadInput3 = IN_3D(k  , new_j, new_i);
    t2_threadInput3 = IN_3D(b  , new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+128)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+128)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    t3_threadInput4 = IN_3D(k  , new_j, new_i);
    t2_threadInput4 = IN_3D(b  , new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+160)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+160)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    t3_threadInput5 = IN_3D(k  , new_j, new_i);
    t2_threadInput5 = IN_3D(b  , new_j, new_i);

#pragma unroll // it seems the loop-unroll is useless to performance
    for(; k < k_end; ++k)
    {
        sum0 = 0.0;
        sum1 = 0.0;
        sum2 = 0.0;
        sum3 = 0.0;
        // move the current storage down 
        t1_threadInput0 = t2_threadInput0;
        t1_threadInput1 = t2_threadInput1;
        t1_threadInput2 = t2_threadInput2;
        t1_threadInput3 = t2_threadInput3;
        t1_threadInput4 = t2_threadInput4;
        t1_threadInput5 = t2_threadInput5;

        t2_threadInput0 = t3_threadInput0;
        t2_threadInput1 = t3_threadInput1;
        t2_threadInput2 = t3_threadInput2;
        t2_threadInput3 = t3_threadInput3;
        t2_threadInput4 = t3_threadInput4;
        t2_threadInput5 = t3_threadInput5;

        int t = (k == Z-1)    ? k : k + 1;
        new_i = (warp_id_x<<3) + lane_id%10-1;  
        new_j = (warp_id_y<<2) + lane_id/10-1;     
        new_j = (new_j == -1)   ? 0   : new_j;
        new_j = (new_j >= M)    ? M-1 : new_j;
        new_i = (new_i == -1)   ? 0   : new_i;
        new_i = (new_i >= N)    ? N-1 : new_i;
        t3_threadInput0 = IN_3D(t  , new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+32)%10-1;
        new_j = (warp_id_y<<2) + (lane_id+32)/10-1;
        new_j = (new_j == -1)   ? 0   : new_j;
        new_j = (new_j >= M)    ? M-1 : new_j;
        new_i = (new_i == -1)   ? 0   : new_i;
        new_i = (new_i >= N)    ? N-1 : new_i;
        t3_threadInput1 = IN_3D(t  , new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+64)%10-1;
        new_j = (warp_id_y<<2) + (lane_id+64)/10-1;
        new_j = (new_j == -1)   ? 0   : new_j;
        new_j = (new_j >= M)    ? M-1 : new_j;
        new_i = (new_i == -1)   ? 0   : new_i;
        new_i = (new_i >= N)    ? N-1 : new_i;
        t3_threadInput2 = IN_3D(t  , new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+96)%10-1;
        new_j = (warp_id_y<<2) + (lane_id+96)/10-1;
        new_j = (new_j == -1)   ? 0   : new_j;
        new_j = (new_j >= M)    ? M-1 : new_j;
        new_i = (new_i == -1)   ? 0   : new_i;
        new_i = (new_i >= N)    ? N-1 : new_i;
        t3_threadInput3 = IN_3D(t  , new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+128)%10-1;
        new_j = (warp_id_y<<2) + (lane_id+128)/10-1;
        new_j = (new_j == -1)   ? 0   : new_j;
        new_j = (new_j >= M)    ? M-1 : new_j;
        new_i = (new_i == -1)   ? 0   : new_i;
        new_i = (new_i >= N)    ? N-1 : new_i;
        t3_threadInput4 = IN_3D(t  , new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+160)%10-1;
        new_j = (warp_id_y<<2) + (lane_id+160)/10-1;
        new_j = (new_j == -1)   ? 0   : new_j;
        new_j = (new_j >= M)    ? M-1 : new_j;
        new_i = (new_i == -1)   ? 0   : new_i;
        new_i = (new_i >= N)    ? N-1 : new_i;
        t3_threadInput5 = IN_3D(t  , new_j, new_i);

        friend_id0 = (lane_id+11+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+19+((lane_id>>3)<<1))&(warpSize-1);
        friend_id2 = (lane_id+27+((lane_id>>3)<<1))&(warpSize-1);
        friend_id3 = (lane_id+3 +((lane_id>>3)<<1))&(warpSize-1);
        tx0 = a0*__shfl(t1_threadInput0, friend_id0);
        ty0 = a0*__shfl(t1_threadInput1, friend_id0);
        tx1 = a0*__shfl(t1_threadInput1, friend_id1);
        ty1 = a0*__shfl(t1_threadInput2, friend_id1);
        tx2 = a0*__shfl(t1_threadInput2, friend_id2);
        ty2 = a0*__shfl(t1_threadInput3, friend_id2);
        tz2 = a0*__shfl(t1_threadInput4, friend_id2);
        tx3 = a0*__shfl(t1_threadInput4, friend_id3);
        ty3 = a0*__shfl(t1_threadInput5, friend_id3);
        sum0 += (lane_id < 17)? tx0: ty0;
        sum1 += (lane_id < 11)? tx1: ty1;
        sum2 += (lane_id < 5 )? tx2: ((lane_id < 31)? ty2: tz2);
        sum3 += (lane_id < 24)? tx3: ty3;
        tx0 = a3*__shfl(t2_threadInput0, friend_id0);
        ty0 = a3*__shfl(t2_threadInput1, friend_id0);
        tx1 = a3*__shfl(t2_threadInput1, friend_id1);
        ty1 = a3*__shfl(t2_threadInput2, friend_id1);
        tx2 = a3*__shfl(t2_threadInput2, friend_id2);
        ty2 = a3*__shfl(t2_threadInput3, friend_id2);
        tz2 = a3*__shfl(t2_threadInput4, friend_id2);
        tx3 = a3*__shfl(t2_threadInput4, friend_id3);
        ty3 = a3*__shfl(t2_threadInput5, friend_id3);
        sum0 += (lane_id < 17)? tx0: ty0;
        sum1 += (lane_id < 11)? tx1: ty1;
        sum2 += (lane_id < 5 )? tx2: ((lane_id < 31)? ty2: tz2);
        sum3 += (lane_id < 24)? tx3: ty3;
        tx0 = a5*__shfl(t3_threadInput0, friend_id0);
        ty0 = a5*__shfl(t3_threadInput1, friend_id0);
        tx1 = a5*__shfl(t3_threadInput1, friend_id1);
        ty1 = a5*__shfl(t3_threadInput2, friend_id1);
        tx2 = a5*__shfl(t3_threadInput2, friend_id2);
        ty2 = a5*__shfl(t3_threadInput3, friend_id2);
        tz2 = a5*__shfl(t3_threadInput4, friend_id2);
        tx3 = a5*__shfl(t3_threadInput4, friend_id3);
        ty3 = a5*__shfl(t3_threadInput5, friend_id3);
        sum0 += (lane_id < 17)? tx0: ty0;
        sum1 += (lane_id < 11)? tx1: ty1;
        sum2 += (lane_id < 5 )? tx2: ((lane_id < 31)? ty2: tz2);
        sum3 += (lane_id < 24)? tx3: ty3;

        friend_id0 = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+9 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id2 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
        friend_id3 = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = a1*__shfl(t2_threadInput0, friend_id0);
        ty0 = a1*__shfl(t2_threadInput1, friend_id0);
        tx1 = a1*__shfl(t2_threadInput1, friend_id1);
        ty1 = a1*__shfl(t2_threadInput2, friend_id1);
        tx2 = a1*__shfl(t3_threadInput2, friend_id2);
        ty2 = a1*__shfl(t3_threadInput3, friend_id2);
        tx3 = a1*__shfl(t3_threadInput3, friend_id3);
        ty3 = a1*__shfl(t3_threadInput4, friend_id3);
        sum0 += (lane_id < 25)? tx0: ty0;
        sum1 += (lane_id < 19)? tx1: ty1;
        sum2 += (lane_id < 13)? tx2: ty2;
        sum3 += (lane_id < 7 )? tx3: ty3;
        friend_id0 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
        friend_id2 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
        friend_id3 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
        tx0 = a2*__shfl(t2_threadInput0, friend_id0);
        ty0 = a2*__shfl(t2_threadInput1, friend_id0);
        tx1 = a2*__shfl(t2_threadInput1, friend_id1);
        ty1 = a2*__shfl(t2_threadInput2, friend_id1);
        tx2 = a2*__shfl(t3_threadInput2, friend_id2);
        ty2 = a2*__shfl(t3_threadInput3, friend_id2);
        tx3 = a2*__shfl(t3_threadInput3, friend_id3);
        ty3 = a2*__shfl(t3_threadInput4, friend_id3);
        sum0 += (lane_id < 18)? tx0: ty0;
        sum1 += (lane_id < 12)? tx1: ty1;
        sum2 += (lane_id < 6 )? tx2: ty2;
        sum3 += (lane_id < 24)? tx3: ty3;
        friend_id0 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
        friend_id2 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
        friend_id3 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
        tx0 = a4*__shfl(t2_threadInput0, friend_id0);
        ty0 = a4*__shfl(t2_threadInput1, friend_id0);
        tx1 = a4*__shfl(t2_threadInput1, friend_id1);
        ty1 = a4*__shfl(t2_threadInput2, friend_id1);
        tx2 = a4*__shfl(t3_threadInput2, friend_id2);
        ty2 = a4*__shfl(t3_threadInput3, friend_id2);
        tz2 = a4*__shfl(t3_threadInput4, friend_id2);
        tx3 = a4*__shfl(t3_threadInput4, friend_id3);
        ty3 = a4*__shfl(t3_threadInput5, friend_id3);
        sum0 += (lane_id < 16)? tx0: ty0;
        sum1 += (lane_id < 10)? tx1: ty1;
        sum2 += (lane_id < 4 )? tx2: ((lane_id < 30)? ty2: tz2);
        sum3 += (lane_id < 24)? tx3: ty3;
        friend_id0 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
        friend_id2 = (lane_id+5 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id3 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = a5*__shfl(t2_threadInput0, friend_id0);
        ty0 = a5*__shfl(t2_threadInput1, friend_id0);
        tx1 = a5*__shfl(t2_threadInput1, friend_id1);
        ty1 = a5*__shfl(t2_threadInput2, friend_id1);
        tz1 = a5*__shfl(t2_threadInput3, friend_id1);
        tx2 = a5*__shfl(t3_threadInput3, friend_id2);
        ty2 = a5*__shfl(t3_threadInput4, friend_id2);
        tx3 = a5*__shfl(t3_threadInput4, friend_id3);
        ty3 = a5*__shfl(t3_threadInput5, friend_id3);
        sum0 += (lane_id < 9 )? tx0: ty0;
        sum1 += (lane_id < 3 )? tx1: ((lane_id < 29)? ty1: tz1);
        sum2 += (lane_id < 23)? tx2: ty2;
        sum3 += (lane_id < 16)? tx3: ty3;

        OUT_3D(k,j  ,i) = sum0;
        OUT_3D(k,j+4,i) = sum1;
        OUT_3D(k,j+8,i) = sum2;
        OUT_3D(k,j+12,i) = sum3;
    }
}

int main(int argc, char **argv)
{
    // int z = 192; // need to be multiple of 64
    // int m = 160;
    // int n = 1600; // need to be mutiple of 64 
#ifdef __DEBUG
    int z = 8;
    int m = 8;
    int n = 8;
#else
    int z = 256; 
    int m = 256;
    int n = 256; 
#endif
    // int z = 192;
    // int m = 160;
    // int n = 1612;
    // int halo = 1;
    // int total = (z+2*halo)*(m+2*halo)*(n+2*halo);
    int total = (z)*(m)*(n);
    const int K = 7;
    DATA_TYPE args[K] = {1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0};
    DATA_TYPE *in = new DATA_TYPE[total];
    DATA_TYPE *out_ref = new DATA_TYPE[total];
    Init_Input_3D(in, z, m, n);

    // Show_Me(in, z, m, n, "Input:");
    for(int i =0; i< ITER; i++)
    {
        Stencil_Seq(in, out_ref, args[0], args[1], args[2], 
            args[3], args[4], args[5], args[6], z, m, n);
        swap(in, out_ref);
    }
    swap(in, out_ref);
    // Show_Me(out_ref, z, m, n, "Output:");


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    float time_wo_pci;

    DATA_TYPE *in_d;
    DATA_TYPE *out_d;
    DATA_TYPE *out = new DATA_TYPE[total];
    hipMalloc((void**)&in_d, total*sizeof(DATA_TYPE));
    hipMalloc((void**)&out_d, total*sizeof(DATA_TYPE));
    Init_Input_3D(in, z, m, n);
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid(n/8, m/4, z/8);
    dim3 dimBlock(8, 4, 8);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda<<<dimGrid, dimBlock>>>(in_d, out_d, args[0], args[1], args[2], 
            args[3], args[4], args[5], args[6], z, m, n); 
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);

    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, z, m, n, "Output(Cuda):");

    cout << "Verify Cuda: " << boolalpha << Verify(out, out_ref, total) << endl;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Cuda Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z, m, n, ITER, 13, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(z, m, n, ITER, time_wo_pci));


    Init_Input_3D(in, z, m, n); // reset input
    Clear_Output_3D(out, z, m, n); // flush output
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, (total)*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid1((n)/64, (m)/4, 4);
    dim3 dimBlock1(64, 4, 1);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda_Sweep<<<dimGrid1, dimBlock1>>>(in_d, out_d, args[0], args[1], args[2], 
            args[3], args[4], args[5], args[6], z, m, n);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
#ifdef __DEBUG_
    Show_Me(out, z, m, n, "Output(Sweep):");
#endif
    cout << "Verify Cuda_Sweep: " << boolalpha << Verify(out, out_ref, total) << endl;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Cuda_Sweep Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z, m, n, ITER, 13, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(z, m, n, ITER, time_wo_pci));


    Init_Input_3D(in, z, m, n); // reset input
    Clear_Output_3D(out, z, m, n); // flush output
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, (total)*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid2((n)/8, (m)/4, (z)/8);
    dim3 dimBlock2(8, 4, 8);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda_Sm<<<dimGrid2, dimBlock2, ((SM_Z)*(SM_M)*(SM_N))*sizeof(DATA_TYPE)>>>(
            in_d, out_d, args[0], args[1], args[2], args[3], args[4], args[5], args[6], z, m, n);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
#ifdef __DEBUG_
    Show_Me(out, z, m, n, "Output(Cuda_Sm):");
#endif
    cout << "Verify Cuda_Sm: " << boolalpha << Verify(out, out_ref, total) << endl;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Cuda_Sm Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z, m, n, ITER, 13, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(z, m, n, ITER, time_wo_pci));


    Init_Input_3D(in, z, m, n); // reset input
    Clear_Output_3D(out, z, m, n); // flush output
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, (total)*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid3((n)/64, (m)/4, 4);
    dim3 dimBlock3(64, 4, 1);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda_Sweep_Sm<<<dimGrid3, dimBlock3, ((SM_2D_M)*(SM_2D_N)*sizeof(DATA_TYPE))>>>(
            in_d, out_d, args[0], args[1], args[2], args[3], args[4], args[5], args[6], z, m, n);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
#ifdef __DEBUG_
    Show_Me(out, z, m, n, "Output(Cuda_Sweep_Sm):");
#endif
    cout << "Verify Cuda_Sweep_Sm: " << boolalpha << Verify(out, out_ref, total) << endl;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Cuda_Sweep_Sm Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z, m, n, ITER, 13, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(z, m, n, ITER, time_wo_pci));


    Init_Input_3D(in, z, m, n); // reset input
    Clear_Output_3D(out, z, m, n); // flush output
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, (total)*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid4((n)/8, (m)/4, (z)/8);
    dim3 dimBlock4(8, 4, 8);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda_Shfl<<<dimGrid4, dimBlock4>>>(
            in_d, out_d, args[0], args[1], args[2], args[3], args[4], args[5], args[6], z, m, n);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
#ifdef __DEBUG
    Show_Me(out, z, m, n, "Output(Cuda_Shfl):");
#endif
    cout << "Verify Cuda_Shfl: " << boolalpha << Verify(out, out_ref, total) << endl;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Cuda_Shfl Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z, m, n, ITER, 13, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(z, m, n, ITER, time_wo_pci));


    Init_Input_3D(in, z, m, n); // reset input
    Clear_Output_3D(out, z, m, n); // flush output
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, (total)*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid5((n)/8, (m)/4, (z)/(8*2));
    dim3 dimBlock5(8, 4, 8);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda_Shfl2<<<dimGrid5, dimBlock5>>>(
            in_d, out_d, args[0], args[1], args[2], args[3], args[4], args[5], args[6], z, m, n);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
#ifdef __DEBUG_
    Show_Me(out, z, m, n, "Output(Cuda_Shfl2):");
#endif
    cout << "Verify Cuda_Shfl2: " << boolalpha << Verify(out, out_ref, total) << endl;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Cuda_Shfl2 Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z, m, n, ITER, 13, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(z, m, n, ITER, time_wo_pci));

    Init_Input_3D(in, z, m, n); // reset input
    Clear_Output_3D(out, z, m, n); // flush output
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, (total)*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid6((n)/8, (m)/4, (z)/(8*4));
    dim3 dimBlock6(8, 4, 8);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda_Shfl4<<<dimGrid6, dimBlock6>>>(
            in_d, out_d, args[0], args[1], args[2], args[3], args[4], args[5], args[6], z, m, n);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
#ifdef __DEBUG_
    Show_Me(out, z, m, n, "Output(Cuda_Shfl4):");
#endif
    cout << "Verify Cuda_Shfl4: " << boolalpha << Verify(out, out_ref, total) << endl;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Cuda_Shfl4 Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z, m, n, ITER, 13, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(z, m, n, ITER, time_wo_pci));


    Init_Input_3D(in, z, m, n); // reset input
    Clear_Output_3D(out, z, m, n); // flush output
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, (total)*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid7((n)/8, (m)/4, (z)/(8*8));
    dim3 dimBlock7(8, 4, 8);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda_Shfl8<<<dimGrid7, dimBlock7>>>(
            in_d, out_d, args[0], args[1], args[2], args[3], args[4], args[5], args[6], z, m, n);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
#ifdef __DEBUG_
    Show_Me(out, z, m, n, "Output(Cuda_Shfl8):");
#endif
    cout << "Verify Cuda_Shfl8: " << boolalpha << Verify(out, out_ref, total) << endl;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Cuda_Shfl8 Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z, m, n, ITER, 13, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(z, m, n, ITER, time_wo_pci));


    Init_Input_3D(in, z, m, n); // reset input
    Clear_Output_3D(out, z, m, n); // flush output
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, (total)*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid8((n)/8, (m)/32, 4);
    dim3 dimBlock8(8, 32, 1);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda_Sweep_Shfl<<<dimGrid8, dimBlock8>>>(
            in_d, out_d, args[0], args[1], args[2], args[3], args[4], args[5], args[6], z, m, n);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
#ifdef __DEBUG_
    Show_Me(out, z, m, n, "Output(Cuda_Sweep_Shfl):");
#endif
    cout << "Verify Cuda_Sweep_Shfl: " << boolalpha << Verify(out, out_ref, total) << endl;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Cuda_Sweep_Shfl Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z, m, n, ITER, 13, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(z, m, n, ITER, time_wo_pci));


    Init_Input_3D(in, z, m, n); // reset input
    Clear_Output_3D(out, z, m, n); // flush output
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, (total)*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid9((n)/8, (m)/(32*2), 4);
    dim3 dimBlock9(8, 32, 1);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda_Sweep_Shfl2<<<dimGrid9, dimBlock9>>>(
            in_d, out_d, args[0], args[1], args[2], args[3], args[4], args[5], args[6], z, m, n);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
#ifdef __DEBUG_
    Show_Me(out, z, m, n, "Output(Cuda_Sweep_Shfl2):");
#endif
    cout << "Verify Cuda_Sweep_Shfl2: " << boolalpha << Verify(out, out_ref, total) << endl;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Cuda_Sweep_Shfl2 Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z, m, n, ITER, 13, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(z, m, n, ITER, time_wo_pci));

    Init_Input_3D(in, z, m, n); // reset input
    Clear_Output_3D(out, z, m, n); // flush output
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, (total)*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid10((n)/8, (m)/(32*4), 4);
    dim3 dimBlock10(8, 32, 1);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda_Sweep_Shfl4<<<dimGrid10, dimBlock10>>>(
            in_d, out_d, args[0], args[1], args[2], args[3], args[4], args[5], args[6], z, m, n);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
#ifdef __DEBUG_
    Show_Me(out, z, m, n, "Output(Cuda_Sweep_Shfl4):");
#endif
    cout << "Verify Cuda_Sweep_Shfl4: " << boolalpha << Verify(out, out_ref, total) << endl;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Cuda_Sweep_Shfl4 Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z, m, n, ITER, 13, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(z, m, n, ITER, time_wo_pci));

    hipFree(in_d);
    hipFree(out_d);


    delete[] in;
    delete[] out;
    delete[] out_ref;

}
